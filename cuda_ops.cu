#include "hip/hip_runtime.h"
#include "cuda_ops.h"
#include <stdio.h>
#include <hipblas.h>

// kernels
namespace kernels {

#if __CUDA_ARCH__ < 600
__device__ double atomicAdd(double* address, double val) {
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
				__double_as_longlong(val + __longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}
#endif

__global__ void compute_weights(float local,
								double * distances,
								unsigned int deformed_points_count,
								unsigned int triangles_count,
								double *weights,
								double *weights_sums) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= deformed_points_count || j >= triangles_count)
		return;
	double distance = distances[i * triangles_count + j];
	weights[i * triangles_count + j] = 1 / (1 + pow(distance, (double)local));
	weights_sums[i] = 0;
};

__global__ void sum_weights(unsigned int deformed_points_count,
							unsigned int triangles_count,
							double *weights,
							double *weights_sums) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= deformed_points_count || j >= triangles_count)
		return;
	atomicAdd(&(weights_sums[i]),
			  weights[i * triangles_count + j]);
};

__global__ void normalise_weights(unsigned int deformed_points_count,
								  unsigned int triangles_count,
								  double *weights,
								  double *weights_sums,
								  double *normalised_weights) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= deformed_points_count || j >= triangles_count)
		return;
	unsigned int offset = i * triangles_count + j;
	normalised_weights[offset] = weights[offset] / weights_sums[i];
};

__device__ double3 double3_sub(double3 a, double3 b) {
	return make_double3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ double3 double3_cross(double3 a, double3 b) {
	return make_double3(a.y * b.z - a.z * b.y,
						a.z * b.x - a.x * b.z,
						a.x * b.y - a.y * b.x);
}

__device__ double double3_dot(double3 a, double3 b) {
	return a.x * b.x + a.y * b.y + a.z * b.z;;
}

__device__ double double3_inv_length(double3 v) {
	return rsqrt(double3_dot(v, v));
}

__device__ double3 double3_double_mult(double3 a, double b) {
    return make_double3(a.x * b, a.y * b, a.z * b);
}

__global__ void create_def_matrices_blas(
			unsigned int triangles_count,
			unsigned int *triangles_indices,
			double *vertices,
			double **res
) {
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= triangles_count)
		return;
	unsigned int index_A = triangles_indices[i * 3] * 3;
	unsigned int index_C = triangles_indices[i * 3 + 1] * 3;
	unsigned int index_B = triangles_indices[i * 3 + 2] * 3;
		
	double3 A = make_double3(
		vertices[index_A],
		vertices[index_A + 1],
		vertices[index_A + 2]
	);
	double3 B = make_double3(
		vertices[index_B],
		vertices[index_B + 1],
		vertices[index_B + 2]
	);
	double3 C = make_double3(
		vertices[index_C],
		vertices[index_C + 1],
		vertices[index_C + 2]
	);
	double3 E1 = double3_sub(C, A);
	double3 E2 = double3_sub(B, A);
	double3 E1_E2_cross = double3_cross(E1, E2);
	double cross_inv_len = double3_inv_length(E1_E2_cross);

	//if (isinf(cross_inv_len)) cross_inv_len = 1;
	double3 E3 = double3_double_mult(E1_E2_cross, cross_inv_len);

	// store matrices in the column-major order
	res[i][0] = E1.x;
	res[i][1] = E1.y;
	res[i][2] = E1.z;
	res[i][3] = 0;
	res[i][4] = E2.x;
	res[i][5] = E2.y;
	res[i][6] = E2.z;
	res[i][7] = 0;
	res[i][8] = E3.x;
	res[i][9] = E3.y;
	res[i][10] = E3.z;
	res[i][11] = 0;
	res[i][12] = 0;
	res[i][13] = 0;
	res[i][14] = 0;
	res[i][15] = 1;
};

__device__ double3 double3_add(double3 a, double3 b) {
	return make_double3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__global__ void apply_deform(unsigned int deformed_points_count,
							 unsigned int triangles_count,
							 double *deformed_points,
							 double **mats,
							 double *cs_pts,
							 double *normalised_weights,
							 double *res_points)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= deformed_points_count || j >= triangles_count)
		return;
	/*
	double3 point = make_double3(
		deformed_points[i],
		deformed_points[i + 1],
		deformed_points[i + 2]
	);
	*/
	unsigned int w_offset = i * triangles_count + j;
	unsigned int c_offset = i * triangles_count * 3 + j * 3;
	double weight = normalised_weights[w_offset];
	double3 cs_point = make_double3(
		cs_pts[c_offset],
		cs_pts[c_offset + 1],
		cs_pts[c_offset + 2]
	);

	// multiply by matrix
	double x = (
		mats[j][0] * cs_point.x +
		mats[j][4] * cs_point.y +
		mats[j][8] * cs_point.z
	);
	double y = (
		mats[j][1] * cs_point.x +
		mats[j][5] * cs_point.y +
		mats[j][9] * cs_point.z
	);
	double z = (
		mats[j][2] * cs_point.x +
		mats[j][6] * cs_point.y +
		mats[j][10] * cs_point.z
	);
	double3 control_point = make_double3(
		x * weight,
		y * weight,
		z * weight
	);

	atomicAdd(&(res_points[i * 3]), control_point.x);
	atomicAdd(&(res_points[i * 3 + 1]), control_point.y);
	atomicAdd(&(res_points[i * 3 + 2]), control_point.z);
};

__global__ void compute_cs_points(unsigned int deformed_points_count,
							 unsigned int triangles_count,
							 double *deformed_points,
							 double **mats,
							 double *cs_pts) {

	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= deformed_points_count || j >= triangles_count)
		return;

	double3 point = make_double3(
		deformed_points[i],
		deformed_points[i + 1],
		deformed_points[i + 2]
	);

	double x = (
		mats[j][0] * point.x +
		mats[j][3] * point.y +
		mats[j][6] * point.z
	);
	double y = (
		mats[j][1] * point.x +
		mats[j][4] * point.y +
		mats[j][7] * point.z
	);
	double z = (
		mats[j][2] * point.x +
		mats[j][5] * point.y +
		mats[j][8] * point.z
	);

	unsigned int c_offset = i * triangles_count * 3 + j * 3;
	cs_pts[c_offset] = x;
	cs_pts[c_offset + 1] = y;
	cs_pts[c_offset + 2] = z;
}


}

// wrappers
int CudaDeviceCount()
{
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	return deviceCount;
};

void CudaComputeWeights(
	double *normalised_weights,
	double *contol_space_points,
	float local,
	double *distances,
	double *points,
	unsigned int deformed_points_count,
	unsigned int triangles_count,
	unsigned int *triangles_indices,
	double *ref_vertices_cu,
	unsigned int ref_vertices_count,
	double *reference_matrices_cu
) {
	hipError_t error;

	unsigned int grid_area = deformed_points_count * triangles_count;
	// internal vars
	double *d_weights_sums;
	error = hipMalloc((void **)&d_weights_sums,
			   sizeof(double) * deformed_points_count);
	hipMemset(d_weights_sums, 0, sizeof(double) * deformed_points_count);

	double *d_weights;
	error = hipMalloc((void **)&d_weights, sizeof(double) * grid_area);
	hipMemset(d_weights, 0, sizeof(double) * grid_area);

	// output vars
	double *d_normalised_weights;
	error = hipMalloc((void **)&d_normalised_weights,
					   sizeof(double) * grid_area);
	hipMemset(d_normalised_weights, 0, sizeof(double) * grid_area);

	// input vars
	double *d_distances;
	error = hipMalloc((void **)&d_distances, sizeof(double) * grid_area);
	error = hipMemcpy(d_distances, distances, sizeof(double) * grid_area,
			   		   hipMemcpyHostToDevice);

	dim3 threadsPerBlock(8, 8);
	dim3 numBlocks(deformed_points_count / threadsPerBlock.x + 1,
				   triangles_count / threadsPerBlock.y + 1);

	
	double *d_points;
	error = hipMalloc((void **)&d_points,
					   sizeof(double) * deformed_points_count * 3);
	error = hipMemcpy(d_points, points,
					   sizeof(double) * deformed_points_count * 3,
					   hipMemcpyHostToDevice);
	if(error != hipSuccess)
		printf("cpy pts: %s\n", hipGetErrorString(error));

	unsigned int *d_triangles_indices;
	error = hipMalloc((void **)&d_triangles_indices,
					   sizeof(unsigned int) * triangles_count * 3);
	error = hipMemcpy(d_triangles_indices, triangles_indices,
					   sizeof(unsigned int) * triangles_count * 3,
					   hipMemcpyHostToDevice);
	if(error != hipSuccess)
		printf("cpy indices: %s\n", hipGetErrorString(error));

	double *d_ref_vertices_cu;
	error = hipMalloc((void **)&d_ref_vertices_cu,
					   sizeof(double) * ref_vertices_count * 3);
	error = hipMemcpy(d_ref_vertices_cu, ref_vertices_cu,
					   sizeof(double) * ref_vertices_count * 3,
					   hipMemcpyHostToDevice);

	double **mats = (double **)malloc(triangles_count * sizeof(double *));
	double **d_mats, *d_mats_flat;
	hipMalloc(&d_mats, triangles_count * sizeof(double *));
	hipMalloc(&d_mats_flat, sizeof(double) * triangles_count * 16);
	mats[0] = d_mats_flat;
	for (int i = 1; i < triangles_count; i++)
		mats[i] = mats[i - 1] + (4 * 4);
	hipMemcpy(d_mats, mats, triangles_count * sizeof(double *),
				hipMemcpyHostToDevice);

	double **mats_inv = (double **)malloc(triangles_count * sizeof(double *));
	double **d_mats_inv, *d_mats_inv_flat;
	hipMalloc(&d_mats_inv, triangles_count * sizeof(double *));
	hipMalloc(&d_mats_inv_flat, sizeof(double) * triangles_count * 16);
	mats_inv[0] = d_mats_inv_flat;
	for (int i = 1; i < triangles_count; i++)
		mats_inv[i] = mats_inv[i - 1] + (4 * 4);
	hipMemcpy(d_mats_inv, mats_inv, triangles_count * sizeof(double *),
				hipMemcpyHostToDevice);

	double *d_cs_points;
	error = hipMalloc((void **)&d_cs_points, sizeof(double) * grid_area * 3);
	hipMemset(d_cs_points, 0, sizeof(double) * grid_area * 3);


	if(error != hipSuccess)
		printf("cpy drv verts: %s\n", hipGetErrorString(error));

	dim3 threadsMatPerBlock(64, 1);
	dim3 numMatBlocks(triangles_count / threadsMatPerBlock.x + 1, 1);
	kernels::create_def_matrices_blas<<<numMatBlocks, threadsMatPerBlock>>>(
		triangles_count,
		d_triangles_indices,
		d_ref_vertices_cu,
		d_mats
	);
	error = hipGetLastError();
	if(error != hipSuccess)
		printf("create_def_matrices: %s\n", hipGetErrorString(error));

	hipblasHandle_t handle;
	hipblasStatus_t status;
   	status = hipblasCreate(&handle);
	if(status != HIPBLAS_STATUS_SUCCESS)
		printf("cublas create: %d\n", status);

    int *info_d;
	hipMalloc(&info_d, triangles_count * sizeof(int));

	status = cublasDmatinvBatched(
		handle, 4,
		(const double**)d_mats, 4,
		d_mats_inv, 4,
		info_d,
		triangles_count
	);
	if(status != HIPBLAS_STATUS_SUCCESS)
		printf("cublas invers: %d\n", status);
	error = hipGetLastError();
	if(error != hipSuccess)
		printf("cuda invers: %s\n", hipGetErrorString(error));

	kernels::compute_cs_points<<<numBlocks, threadsPerBlock>>>(
		deformed_points_count, triangles_count, d_points,
		d_mats_inv, d_cs_points
	);
	error = hipGetLastError();
	if(error != hipSuccess)
		printf("compute_cs_points: %s\n", hipGetErrorString(error));
	
	kernels::compute_weights<<<numBlocks, threadsPerBlock>>>(
		local, d_distances, deformed_points_count, triangles_count, d_weights, d_weights_sums
	);
	error = hipGetLastError();
	if(error != hipSuccess)
		printf("compute_weights: %s\n", hipGetErrorString(error));

	error = hipGetLastError();
	if(error != hipSuccess)
		printf("Compute weights: %s\n", hipGetErrorString(error));
	
	kernels::sum_weights<<<numBlocks, threadsPerBlock>>>(
		deformed_points_count, triangles_count, d_weights, d_weights_sums
	);

	error = hipGetLastError();
	if(error != hipSuccess)
		printf("Sum weights: %s\n", hipGetErrorString(error));

	kernels::normalise_weights<<<numBlocks, threadsPerBlock>>>(
		deformed_points_count, triangles_count, d_weights,
		d_weights_sums, d_normalised_weights 
	);

	if(error != hipSuccess)
		printf("Normalise weights: %s\n", hipGetErrorString(error));

	error = hipGetLastError();

	error = hipMemcpy(normalised_weights, d_normalised_weights,
			   sizeof(double) * deformed_points_count * triangles_count,
			   hipMemcpyDeviceToHost);

	error = hipMemcpy(contol_space_points, d_cs_points,
			   sizeof(double) * grid_area * 3,
			   hipMemcpyDeviceToHost);

	if(error != hipSuccess)
		printf("MemCpy DevToHost: %s\n", hipGetErrorString(error));

	hipFree(d_distances);
	hipFree(d_weights);
	hipFree(d_weights_sums);
	hipFree(d_normalised_weights);
	hipFree(d_points);
	hipFree(d_triangles_indices);
	hipFree(d_ref_vertices_cu);
	hipFree(d_mats);
	hipFree(d_mats_flat);
	hipFree(d_mats_inv);
	hipFree(d_mats_inv_flat);
	hipFree(d_cs_points);
	free(mats);
	free(mats_inv);
};


void CudaApplyDeform(
	double *out_points,
	double *cs_points,
	double *points,
	unsigned int deformed_points_count,
	unsigned int *triangles_indices,
	unsigned int triangles_count,
	double *driver_vertices_cu,
	unsigned int driver_vertices_count,
	double *normalised_weights,
	double *tmats
) {
	hipError_t error;
	unsigned int grid_area = deformed_points_count * triangles_count;
	
	// out vars
	double *d_out_points;
	error = hipMalloc((void **)&d_out_points,
			   sizeof(double) * deformed_points_count * 3);
	hipMemset(d_out_points, 0, sizeof(double) * deformed_points_count * 3);
	if(error != hipSuccess)
		printf("outs: %s\n", hipGetErrorString(error));

	double **mats = (double **)malloc(triangles_count * sizeof(double *));
	double **d_mats, *d_mats_flat;
	hipMalloc(&d_mats, triangles_count * sizeof(double *));
	hipMalloc(&d_mats_flat, sizeof(double) * triangles_count * 16);
	mats[0] = d_mats_flat;
	for (int i = 1; i < triangles_count; i++)
		mats[i] = mats[i - 1] + (4 * 4);
	hipMemcpy(d_mats, mats, triangles_count * sizeof(double *),
				hipMemcpyHostToDevice);

	// in vars
	double *d_cs_points;
	error = hipMalloc((void **)&d_cs_points, sizeof(double) * grid_area * 3);
	error = hipMemcpy(d_cs_points, cs_points, sizeof(double) * grid_area * 3,
					   hipMemcpyHostToDevice);
	double *d_points;
	error = hipMalloc((void **)&d_points,
					   sizeof(double) * deformed_points_count * 3);
	error = hipMemcpy(d_points, points,
					   sizeof(double) * deformed_points_count * 3,
					   hipMemcpyHostToDevice);
	if(error != hipSuccess)
		printf("cpy pts: %s\n", hipGetErrorString(error));
	
	double *d_normalised_weights;
	error = hipMalloc((void **)&d_normalised_weights,
					   sizeof(double) * grid_area);
	error = hipMemcpy(d_normalised_weights, normalised_weights,
					   sizeof(double) * grid_area,
					   hipMemcpyHostToDevice);
	if(error != hipSuccess)
		printf("cpy norm weights: %s\n", hipGetErrorString(error));

	unsigned int *d_triangles_indices;
	error = hipMalloc((void **)&d_triangles_indices,
					   sizeof(unsigned int) * triangles_count * 3);
	error = hipMemcpy(d_triangles_indices, triangles_indices,
					   sizeof(unsigned int) * triangles_count * 3,
					   hipMemcpyHostToDevice);
	if(error != hipSuccess)
		printf("cpy indices: %s\n", hipGetErrorString(error));

	double *d_driver_vertices_cu;
	error = hipMalloc((void **)&d_driver_vertices_cu,
					   sizeof(double) * driver_vertices_count * 3);
	error = hipMemcpy(d_driver_vertices_cu, driver_vertices_cu,
					   sizeof(double) * driver_vertices_count * 3,
					   hipMemcpyHostToDevice);
	if(error != hipSuccess)
		printf("cpy drv verts: %s\n", hipGetErrorString(error));
	
	dim3 threadsMatPerBlock(64, 1);
	dim3 numMatBlocks(triangles_count / threadsMatPerBlock.x + 1, 1);

	kernels::create_def_matrices_blas<<<numMatBlocks, threadsMatPerBlock>>>(
		triangles_count,
		d_triangles_indices,
		d_driver_vertices_cu,
		d_mats
	);

	error = hipGetLastError();
	if(error != hipSuccess)
		printf("Create def mats: %s\n", hipGetErrorString(error));

	dim3 threadsPerBlock(8, 8);
	dim3 numBlocks(deformed_points_count / threadsPerBlock.x + 1,
				   triangles_count / threadsPerBlock.y + 1);

	kernels::apply_deform<<<numBlocks, threadsPerBlock>>>(
		deformed_points_count,
		triangles_count,
		d_points,
		d_mats,
		d_cs_points,
		d_normalised_weights,
		d_out_points
	);
	error = hipGetLastError();
	if(error != hipSuccess)
		printf("Apply deform: %s\n", hipGetErrorString(error));

	error = hipMemcpy(out_points, d_out_points,
			   sizeof(double) * deformed_points_count * 3,
			   hipMemcpyDeviceToHost);

	if(error != hipSuccess)
		printf("Deform DevToHost: %s\n", hipGetErrorString(error));

	hipFree(d_out_points);
	hipFree(d_mats);
	hipFree(d_mats_flat);
	hipFree(d_cs_points);
	hipFree(d_points);
	hipFree(d_triangles_indices);
	hipFree(d_driver_vertices_cu);
	hipFree(d_normalised_weights);
	free(mats);
};
